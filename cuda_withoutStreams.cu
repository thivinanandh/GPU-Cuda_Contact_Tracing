#include "hip/hip_runtime.h"
#include<iostream>
#include<cstring>
#include<fstream>
#include<vector>
#include<sstream>
#include<cstdlib>
#include<algorithm>
#include<omp.h>
#include<random>
#include<ctime>
#include <iomanip>      // std::setw
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


static int N_THREADS;

int static fileShifter(int startLine, const char* filename, int threadNo)
{
    std::string file(filename);
    std::string command ;
    std::string fileout;
    std::string fileoutrm;
    fileout = "b_" + std::to_string(threadNo);
    fileoutrm = "rm " + fileout;
    command  = "head -n " + std::to_string(startLine) + " " + file + " | wc -c >> " + fileout;
    // std::cout << command << std::endl;
    
    system(command.c_str());
    int position;
    std::ifstream file1(fileout.c_str());
    file1 >> position;
    file1.close();
     system(fileoutrm.c_str());
    return position;
}

#include"sparse_parallel.h"

using namespace std;


__global__ void GPU_contactTracer(int* rowPtrDevice,int* colPtrDevice,int* infectedListDevice,int* quarantinedFlagDevice,
                                 int rowSize, int N_threadCycle , int infectedListSize , int quarantinedFlagSize , int day)
{
    const unsigned int ThreadId = blockIdx.x*blockDim.x + threadIdx.x;

    // if(!ThreadId)
    // {
    //     printf(" day : %d\n", day);

    //     for(int k = 0 ; k < infectedListSize ; k++)
    //     {
    //         printf("%d ", infectedListDevice[k]);
    //     }
    // }

    for (int ThreadCycle = 0 ; ThreadCycle < N_threadCycle  ; ThreadCycle++)
    {
        unsigned long int ThreadId_new = ThreadId + ThreadCycle*blockDim.x;             

        if(ThreadId_new < infectedListSize)
        {
            int searchVertex    = infectedListDevice[ThreadId_new];
            int begin           = rowPtrDevice[searchVertex];
            int end             = rowPtrDevice[searchVertex+1];
            int size            = end - begin;

            for (int k = begin; k < end; k++)
            {
                int vertNo = colPtrDevice[k];
                quarantinedFlagDevice[vertNo] = day+1;
            }
        }
    }

}


int main(int argc , char** argv)
{
    if(argc < 3)
    {
        cout << " ERROR : Insufficient input variables  " <<endl;
        cout << " INFO  :  arg-1 - InputFile Name "  <<endl;
        exit(0);   // EXIT Statement
    }
    N_THREADS = stoi(argv[2]);
    srand(time(NULL));

    // ---------------- MAIN FILE VARIABLES ----------------------- //
    //Variable to Store all the filenames  ---1st one - input data , remaining files
    std::vector<string> fileNames;
    std::vector<int> infectedList;          // infected People list
    std::vector<int> quarantinedList;
    std::vector<int> quarantinedFlag;
    int N_inputFiles;

    const int N_STAGE_QUARANTINE = 1;
    const double PROBABILITY_INFECTION = 0.5;
    // --------------------------- END - MAIN FILE VARIABLE --------//
    ifstream file(argv[1]);
    
    //Temp variables 
    std:string temp;
    int t_n ;
    

    // ------ Read the Filenames of the Graph from the Input File ----- //
    while(getline(file,temp))
    {
       t_n = stoi(temp);
        break;    
    }

    fileNames.resize(t_n); 
    N_inputFiles = t_n - 1;

    file.clear(); file.seekg(0);

    int t_nLines = 0;
    while(getline(file,temp))
    {
        if(t_nLines)
            fileNames[t_nLines-1] = temp;

        t_nLines++;
    }

    file.close();
    
    // ------ END Read the Filenames of the Graph from the Input File ----- //

    //------ read input infected file ------- //
    file.open(fileNames[0]);

    while(getline(file,temp))
        infectedList.push_back(stoi(temp));
    
    file.close();
    //------ END read input infected file ------- //

    // create an object for SparseMatrix based on N input files
    SparseMatrix** Graph = new SparseMatrix*[N_inputFiles];

    int maxRow = 0;
    for (int i = 0; i < N_inputFiles ; i++)
    {
        Graph[i] = new SparseMatrix;
        const char* file = fileNames[i + 1].c_str();
        Graph[i]->SparseMatrixRead1(file);

        if (Graph[i]->m_Nrow > maxRow) maxRow = Graph[i]->m_Nrow;

    }

    struct timespec start,end, TotalStart, TotalEnd;
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&start);
    // Graph[1]->printArray();
    // fill the infected list flag array
    int vertices =  maxRow;
    quarantinedFlag.resize(vertices);

    std::vector<int> quarantinedFlag_sol(vertices,0);

    for (int k = 0; k < infectedList.size(); k++)
        quarantinedFlag[infectedList[k]] = 1;


    //Pragma omp shared Variables
    std::vector<int> o_N_infected(N_THREADS,0);
    std::vector<int> o_N_quarantined(N_THREADS,0);
    std::vector<int> o_N_safe(N_THREADS,0);

    
    const int* rowPtr;
    const int* colPtr;
    const double* values;

    int* quarantinedFlagDevice;
    int* infectedListDevice ; 
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&TotalStart);

    hipMalloc((void **) &quarantinedFlagDevice, sizeof(int)*vertices);
    hipMalloc((void **) &infectedListDevice, sizeof(int)*vertices);


    hipMemcpy(quarantinedFlagDevice,quarantinedFlag.data(),sizeof(int)*vertices,hipMemcpyHostToDevice);

    int InfectednewSize  = infectedList.size();
    int prevSize = 0;
    omp_set_num_threads(N_THREADS);


    double TotalTime = 0;
    
    for (int day = 0; day < N_inputFiles; day++)
    {
        double t1 = omp_get_wtime();


        colPtr = Graph[day]->ColPtr.data();
        rowPtr = Graph[day]->RowPtr.data();
        values = Graph[day]->values.data();

        int StartPos = 0;
        int Max_contact         = 0;
        int Max_contact_vertex  = 0;

        for (int stage = 0; stage < N_STAGE_QUARANTINE ; stage++)
        {
            int* rowPtrDevice ;
            int* colPtrDevice ;
            int sizeRowPtr      = Graph[day]->RowPtr.size();
            int sizeColPtr      = Graph[day]->ColPtr.size();
            int sizeInfected   = infectedList.size();

            hipMalloc((void **) &rowPtrDevice, sizeof(int)* sizeRowPtr);
            hipMalloc((void **) &colPtrDevice, sizeof(int)*sizeColPtr);
            cout<< "- New size  : " << InfectednewSize << " Prev size : " << prevSize <<endl;
            hipMemcpy(rowPtrDevice,rowPtr,sizeof(int)* sizeRowPtr,hipMemcpyHostToDevice);
            hipMemcpy(colPtrDevice,colPtr,sizeof(int)*sizeColPtr,hipMemcpyHostToDevice);
            hipMemcpy(infectedListDevice + prevSize,infectedList.data() + prevSize,sizeof(int)*InfectednewSize,hipMemcpyHostToDevice);

            prevSize = sizeInfected;
            
            int numBlocks = 1;
            
            int numThreadsPerBlock;


            if(sizeInfected < 1024) numThreadsPerBlock = sizeInfected;
            else numThreadsPerBlock = 1024;

            int N_threadCycle =  std::ceil( double(sizeInfected)/double(numThreadsPerBlock));
            dim3 dimGrid(numBlocks);
            dim3 dimBlock(numThreadsPerBlock);

            GPU_contactTracer<<<dimGrid,numThreadsPerBlock>>>(rowPtrDevice,colPtrDevice,infectedListDevice,quarantinedFlagDevice,
                                                              sizeRowPtr,N_threadCycle,sizeInfected,vertices , day);
            // hipDeviceSynchronize();

            hipMemcpy(quarantinedFlag_sol.data(),quarantinedFlagDevice,sizeof(int)*vertices,hipMemcpyDeviceToHost);
            
            hipDeviceSynchronize();
            

            InfectednewSize = 0;
            for(int j = 0 ; j < quarantinedFlag_sol.size(); j++)
            {
                if(quarantinedFlag[j] == -99 ) 
                {
                    quarantinedFlag_sol[j] = -99;
                    continue;
                }
                if((quarantinedFlag_sol[j] ==  day+1)  && !abs(quarantinedFlag[j]) )
                {

                    infectedList.push_back(j);
                    InfectednewSize += 1;
                    double randNo = double (rand())/RAND_MAX;
                    if(randNo > PROBABILITY_INFECTION)
                        quarantinedFlag_sol[j]  = -99;
               
                }

                quarantinedFlag[j] = quarantinedFlag_sol[j];
            }
            hipFree(rowPtrDevice);
            hipFree(colPtrDevice);
            // hipFree(infectedListDevice);
            
        }
        

        int N_inf = 0 ;
        int N_quar = 0;
        int N_safe = 0;
        for (int i = 0; i < quarantinedFlag_sol.size(); i++)
        {
            if(quarantinedFlag_sol[i] == -99) N_inf++;
            else if  (quarantinedFlag_sol[i] >= 1) N_quar++;
            else N_safe++;
        }



        double t2 = omp_get_wtime();
        TotalTime += t2 - t1;

        
        int total = quarantinedFlag_sol.size();
        std::cout  << std::endl;
        std::cout << "*************** DAY :  " << N_inputFiles -  day << " *************** " <<std::endl;
        std::cout << "TOTAL PEOPLE  :   "  << vertices << std::endl;
        std::cout << "Max Contact Vertex: " <<Max_contact_vertex << "  Contact Count : " << Max_contact << std::endl;
        std::cout << "---------- QUARANTINED INFORMATION ------------ " << std::endl;
        std::cout <<std::setw(28)<< "INFECTED IN QUARANTINE : " <<std::setw(8) << N_inf <<std::setw(35)<< "Percentage : " << double(N_inf)/total* 100 << " %"<< std::endl;
        std::cout <<std::setw(28)<< "NOT INFECTED IN QUARANTINE : " <<std::setw(8) << N_quar<<std::setw(35)<< "Percentage : " << (double(N_quar)/total) * 100 << " %" << std::endl;
        std::cout <<std::setw(28)<< "TOTAL QUARANTINE : "  <<std::setw(8)<< N_quar + N_inf<<std::setw(35)<< "Total INfected Percentage : " << double((N_quar + N_inf))/total * 100 << " %"    << std::endl;
        std::cout <<std::setw(28)<< "TOTAL SAFE : " <<std::setw(8)<< N_safe  <<std::setw(35)<< "Total Safe Percentage : " << double(N_safe)/total * 100 << " %" << std::endl;
        std::cout <<std::setw(28)<< " ------------ TIME FOR DAY  : " <<  (end.tv_sec - start.tv_sec) + (end.tv_nsec -  start.tv_nsec)*1e-9 <<std::endl;

    }
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&TotalEnd);
    cout << " Time for Iteration : " << TotalTime <<endl;




    return 0;
}
