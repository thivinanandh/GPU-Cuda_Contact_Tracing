#include "hip/hip_runtime.h"
#include<iostream>
#include<cstring>
#include<fstream>
#include<vector>
#include<sstream>
#include<cstdlib>
#include<algorithm>
#include<omp.h>
#include<random>
#include<ctime>
#include <iomanip>      // std::setw
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


static int N_THREADS;

int static fileShifter(int startLine, const char* filename, int threadNo)
{
    std::string file(filename);
    std::string command ;
    std::string fileout;
    std::string fileoutrm;
    fileout = "b_" + std::to_string(threadNo);
    fileoutrm = "rm " + fileout;
    command  = "head -n " + std::to_string(startLine) + " " + file + " | wc -c >> " + fileout;
    // std::cout << command << std::endl;
    
    system(command.c_str());
    int position;
    std::ifstream file1(fileout.c_str());
    file1 >> position;
    file1.close();
     system(fileoutrm.c_str());
    return position;
}

#include"sparse_parallel.h"

using namespace std;


__global__ void GPU_contactTracer(int* rowPtrDevice,int* colPtrDevice,int* infectedListDevice,int* quarantinedFlagDevice,
                                 int N_threadCycle , int infectedListSize , int maxVertSize , int day)
{
    const unsigned int ThreadId = blockIdx.x*blockDim.x + threadIdx.x;

    // if(!ThreadId) printf("---- HPU : infectedliest : %d \n" , infectedListSize);

    for (int ThreadCycle = 0 ; ThreadCycle < N_threadCycle  ; ThreadCycle++)
    {
        unsigned long int ThreadId_new = ThreadId + ThreadCycle*blockDim.x;             

        if(ThreadId_new < infectedListSize)
        {
            int searchVertex    = infectedListDevice[ThreadId_new];
            int begin           = rowPtrDevice[searchVertex];
            int end             = rowPtrDevice[searchVertex+1];
            int size            = end - begin;

            // if(size > 30) end = begin+30;

            for (int k = begin; k < end; k++)
            {
                int vertNo = colPtrDevice[k];
                quarantinedFlagDevice[vertNo] = day+1;
            }
        }
    }

}


int main(int argc , char** argv)
{
    if(argc < 3)
    {
        cout << " ERROR : Insufficient input variables  " <<endl;
        cout << " INFO  :  arg-1 - InputFile Name "  <<endl;
        exit(0);   // EXIT Statement
    }
    N_THREADS = stoi(argv[2]);
    srand(time(NULL));

    // ---------------- MAIN FILE VARIABLES ----------------------- //
    //Variable to Store all the filenames  ---1st one - input data , remaining files
    std::vector<string> fileNames;
    std::vector<int> infectedList;          // infected People list
    std::vector<int> quarantinedList;
    std::vector<int> quarantinedFlag;
    int N_inputFiles;

    const int N_STAGE_QUARANTINE = 1;
    const double PROBABILITY_INFECTION = 0.5;
    // --------------------------- END - MAIN FILE VARIABLE --------//
    ifstream file(argv[1]);
    
    //Temp variables 
    std:string temp;
    int t_n ;
    

    // ------ Read the Filenames of the Graph from the Input File ----- //
    while(getline(file,temp))
    {
       t_n = stoi(temp);
        break;    
    }

    fileNames.resize(t_n); 
    N_inputFiles = t_n - 1;

    file.clear(); file.seekg(0);

    int t_nLines = 0;
    while(getline(file,temp))
    {
        if(t_nLines)
            fileNames[t_nLines-1] = temp;

        t_nLines++;
    }

    file.close();
    
    // ------ END Read the Filenames of the Graph from the Input File ----- //

    //------ read input infected file ------- //
    file.open(fileNames[0]);

    while(getline(file,temp))
        infectedList.push_back(stoi(temp));
    
    file.close();
    //------ END read input infected file ------- //

    // create an object for SparseMatrix based on N input files
    SparseMatrix** Graph = new SparseMatrix*[N_inputFiles];

    int maxRow = 0;
    for (int i = 0; i < N_inputFiles ; i++)
    {
        Graph[i] = new SparseMatrix;
        const char* file = fileNames[i + 1].c_str();
        Graph[i]->SparseMatrixRead1(file);

        if (Graph[i]->m_Nrow > maxRow) maxRow = Graph[i]->m_Nrow;

    }

    struct timespec start,end, TotalStart, TotalEnd;
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&start);
    // Graph[1]->printArray();
    // fill the infected list flag array
    int vertices =  maxRow;
    quarantinedFlag.resize(vertices);

    std::vector<int> quarantinedFlag_sol(vertices,0);

    // std::vector<int> quarDate(N_inputFiles);

    for (int k = 0; k < infectedList.size(); k++)
        quarantinedFlag[infectedList[k]] = 1;

    
    const int* rowPtr;
    const int* colPtr;
    const double* values;

    int* quarantinedFlagDevice;
    int* infectedListDevice ; 
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&TotalStart);

    hipMalloc((void **) &quarantinedFlagDevice, sizeof(int)*vertices);
    hipMalloc((void **) &infectedListDevice, sizeof(int)*vertices);



    // Create Arrays for the Data Transfer
    int** rowPtrDevice = new int*[N_inputFiles] ;
    int** colPtrDevice  = new int*[N_inputFiles];

    // ---------------- Create CUDA Streams --------------------------- //
    hipStream_t stream1[N_inputFiles];
    hipStream_t test;
    hipError_t result[N_inputFiles];

    for (int i = 0 ; i < N_inputFiles ;i++)
        result[i] = hipStreamCreate(&stream1[i]);

    int rowSize = Graph[0]->RowPtr.size();
    int colSize = Graph[0]->ColPtr.size();
 
    int* rowP  = Graph[0]->RowPtr.data() ;
    int* colP = Graph[0]->ColPtr.data();

    hipMalloc((void **) &rowPtrDevice[0], sizeof(int) * rowSize );
    hipMalloc((void **) &colPtrDevice[0], sizeof(int) *  colSize);

    // --- Async Copy of graph Data of day 0 ---- //

    hipMemcpyAsync(rowPtrDevice[0], rowP , sizeof(int) * rowSize, hipMemcpyHostToDevice, stream1[0]);
    hipMemcpyAsync(colPtrDevice[0], colP , sizeof(int) * colSize, hipMemcpyHostToDevice, stream1[0]);
    hipMemcpyAsync(infectedListDevice , infectedList.data()  , sizeof(int)* (infectedList.size()), hipMemcpyHostToDevice, stream1[0]);
    hipMemcpyAsync(quarantinedFlagDevice,quarantinedFlag.data(),sizeof(int)*vertices,hipMemcpyHostToDevice,stream1[0]);

    // pre assignment variablees for Looping
    int InfectednewSize = 0;

    double TotalTime = 0;

    for (int day = 0; day < N_inputFiles; day++)
    {
        clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&start);

        int sizeInfected   = infectedList.size();
        
        int numBlocks = 1;
        
        int numThreadsPerBlock;

        if(sizeInfected < 1024) numThreadsPerBlock = sizeInfected;
        else numThreadsPerBlock = 1024;

        int N_threadCycle =  std::ceil( double(sizeInfected)/double(numThreadsPerBlock));
        dim3 dimGrid(numBlocks);
        dim3 dimBlock(numThreadsPerBlock);

        // --- Async Kernel Call -- //
        
        GPU_contactTracer<<<dimGrid,numThreadsPerBlock,0 , stream1[day]>>>(rowPtrDevice[day],colPtrDevice[day],infectedListDevice,quarantinedFlagDevice,
                                                            N_threadCycle,sizeInfected,vertices , day);

        // --- Async Data Transfer for Day - N+1 , which is Overlapped with the previous Kernel Execution --- // 
        if(day != N_inputFiles-1)
        {
            colPtr = Graph[day+1]->ColPtr.data();
            rowPtr = Graph[day+1]->RowPtr.data();

            int sizeRowPtr      = Graph[day+1]->RowPtr.size();
            int sizeColPtr      = Graph[day+1]->ColPtr.size();

            hipMalloc((void **) &rowPtrDevice[day+1], sizeof(int)*sizeRowPtr);
            hipMalloc((void **) &colPtrDevice[day+1], sizeof(int)*sizeColPtr);
            // cout<< "- New size  : " << InfectednewSize << " Prev size : " << prevSize <<endl;
            hipMemcpyAsync(rowPtrDevice[day+1],rowPtr,sizeof(int)* sizeRowPtr,hipMemcpyHostToDevice,stream1[day+1]);
            hipMemcpyAsync(colPtrDevice[day+1],colPtr,sizeof(int)*sizeColPtr,hipMemcpyHostToDevice,stream1[day+1]);
        }

        // ----- copy the Rsult from kernel for day "N" and Synchronise the stream for day "N" -------- //
        hipMemcpyAsync(quarantinedFlag_sol.data(),quarantinedFlagDevice,sizeof(int)*vertices,hipMemcpyDeviceToHost,stream1[day] );
        hipStreamSynchronize(stream1[day]);
        // hipDeviceSynchronize();
        
        InfectednewSize = 0;
        for(int j = 0 ; j < quarantinedFlag_sol.size(); j++)
        {
            if(quarantinedFlag[j] == -99 ) 
            {
                quarantinedFlag_sol[j] = -99;
                continue;
            }
            if((quarantinedFlag_sol[j] ==  day+1)  && !abs(quarantinedFlag[j]) )
            {

                infectedList.push_back(j);
                InfectednewSize += 1;
                // Mark vertices as infected based on rand number
                double randNo = double (rand())/RAND_MAX;
                if(randNo > PROBABILITY_INFECTION)
                    quarantinedFlag_sol[j]  = -99;
            }

            quarantinedFlag[j] = quarantinedFlag_sol[j];
        }

        // -- Async send the updated infected list ( only newly added on Nth day ) to Device --- //
        if(day != N_inputFiles-1)
        {
         hipMemcpyAsync(infectedListDevice + sizeInfected,infectedList.data() + sizeInfected,
                            sizeof(int)*InfectednewSize,hipMemcpyHostToDevice,stream1[day+1]);
        }

        // Free the allocated memory on device for Day N
        hipFree(rowPtrDevice[day]);
        hipFree(colPtrDevice[day]);

        
        int N_inf = 0 ;
        int N_quar = 0;
        int N_safe = 0;
        
        // #pragma omp parallel for shared(quarantinedFlag_sol,N_inf,N_quar,N_safe)  
        //  -- pragma commented out as it is creating additional over head and increasng execution time 
        

        for (int i = 0; i < quarantinedFlag_sol.size(); i++)
        {
            if(quarantinedFlag_sol[i] == -99) N_inf++;
            else if  (quarantinedFlag_sol[i] >= 1) N_quar++;
            else N_safe++;
        }

        clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&end);

        TotalTime += (end.tv_sec - start.tv_sec) + (end.tv_nsec -  start.tv_nsec)*1e-9;
        
        int total = quarantinedFlag_sol.size();
        std::cout  << std::endl;
        std::cout << "*************** DAY :  " << N_inputFiles -  day << " *************** " <<std::endl;
        std::cout << "TOTAL PEOPLE  :   "  << vertices << std::endl;
        // std::cout << "Max Contact Vertex: " <<Max_contact_vertex << "  Contact Count : " << Max_contact << std::endl;
        std::cout << "---------- QUARANTINED INFORMATION ------------ " << std::endl;
        std::cout <<std::setw(28)<< "INFECTED IN QUARANTINE : " <<std::setw(8) << N_inf <<std::setw(35)<< "Percentage : " << double(N_inf)/total* 100 << " %"<< std::endl;
        std::cout <<std::setw(28)<< "NOT INFECTED IN QUARANTINE : " <<std::setw(8) << N_quar<<std::setw(35)<< "Percentage : " << (double(N_quar)/total) * 100 << " %" << std::endl;
        std::cout <<std::setw(28)<< "TOTAL QUARANTINE : "  <<std::setw(8)<< N_quar + N_inf<<std::setw(35)<< "Total INfected Percentage : " << double((N_quar + N_inf))/total * 100 << " %"    << std::endl;
        std::cout <<std::setw(28)<< "TOTAL SAFE : " <<std::setw(8)<< N_safe  <<std::setw(35)<< "Total Safe Percentage : " << double(N_safe)/total * 100 << " %" << std::endl;
        std::cout <<std::setw(28)<< " ------------ TIME FOR DAY  : " <<  (end.tv_sec - start.tv_sec) + (end.tv_nsec -  start.tv_nsec)*1e-9 <<std::endl;

    }
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID,&TotalEnd);
    cout << " Time for Iteration : " << TotalTime <<endl;




    hipDeviceReset();

    return 0;
}